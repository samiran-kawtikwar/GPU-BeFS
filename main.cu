#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <vector>
#include "heap/bheap.cuh"
#include "utils/logger.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/timer.h"
#include "queue/queue.cuh"
#include "request_manager.cuh"
#include "memory_manager.cuh"
#include "defs.cuh"
#include "LAP/device_utils.cuh"
#include "LAP/Hung_lap.cuh"
#include "LAP/lap_kernels.cuh"
#include "branch.cuh"

#include "RCAP/config.h"
#include "RCAP/cost_generator.h"
#include "RCAP/gurobi_solver.h"

__global__ void get_exit_code(ExitCode *ec)
{

  ec[0] = opt_reached.load(cuda::memory_order_consume)     ? ExitCode::OPTIMAL
          : heap_overflow.load(cuda::memory_order_consume) ? ExitCode::HEAP_FULL
                                                           : ExitCode::UNKNOWN_ERROR;
}

int main(int argc, char **argv)
{
  Log(info, "Starting program");
  Config config = parseArgs(argc, argv);
  printConfig(config);
  int dev_ = config.deviceId;
  uint psize = config.user_n;
  uint ncommodities = config.user_ncommodities;
  if (psize > 100)
  {
    Log(critical, "Problem size too large, Implementation not ready yet. Use problem size <= 100");
    exit(-1);
  }
  CUDA_RUNTIME(hipDeviceReset());
  CUDA_RUNTIME(hipSetDevice(dev_));
  problem_info *h_problem_info = generate_problem<cost_type>(config, config.seed);

  /*
    Log(info, "Costs: ");
    for (size_t i = 0; i < psize; i++)
    {
      for (size_t j = 0; j < psize; j++)
      {
        printf("%u, ", h_problem_info->costs[i * psize + j]);
      }
      printf("\n");
    }

    Log(info, "Weights: ");
    for (size_t k = 0; k < ncommodities; k++)
    {
      printf("Commodity: %lu\n", k);
      for (size_t i = 0; i < psize; i++)
      {
        for (size_t j = 0; j < psize; j++)
        {
          printf("%u, ", h_problem_info->weights[k * psize * psize + i * psize + j]);
        }
        printf("\n");
      }
      printf("\n");
    }

    Log(info, "Budgets: ");
    for (size_t k = 0; k < ncommodities; k++)
    {
      printf("%u, ", h_problem_info->budgets[k]);
    }
    printf("\n");
  */
  // Copy problem info to device
  problem_info *d_problem_info;
  CUDA_RUNTIME(hipMallocManaged((void **)&d_problem_info, sizeof(problem_info)));
  d_problem_info->psize = psize;
  d_problem_info->ncommodities = ncommodities;
  CUDA_RUNTIME(hipMalloc((void **)&d_problem_info->costs, psize * psize * sizeof(cost_type)));
  CUDA_RUNTIME(hipMemcpy(d_problem_info->costs, h_problem_info->costs, psize * psize * sizeof(cost_type), hipMemcpyHostToDevice));
  CUDA_RUNTIME(hipMalloc((void **)&d_problem_info->weights, ncommodities * psize * psize * sizeof(weight_type)));
  CUDA_RUNTIME(hipMemcpy(d_problem_info->weights, h_problem_info->weights, ncommodities * psize * psize * sizeof(weight_type), hipMemcpyHostToDevice));
  CUDA_RUNTIME(hipMalloc((void **)&d_problem_info->budgets, ncommodities * sizeof(weight_type)));
  CUDA_RUNTIME(hipMemcpy(d_problem_info->budgets, h_problem_info->budgets, ncommodities * sizeof(weight_type), hipMemcpyHostToDevice));

  // Solve RCAP
  const cost_type UB = solve_with_gurobi<cost_type, weight_type>(h_problem_info->costs, h_problem_info->weights, h_problem_info->budgets, psize, ncommodities);

  // Log(info, "RCAP solved succesfully, objective %u\n", (uint)UB);
  // printf("Exiting...\n");
  // exit(0);

  Log(debug, "Solving RCAP with Branching");
  Timer t = Timer();

  size_t free, total;
  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %f %", ((total - free) * 1.0) / total * 100);

  // Create space for queue
  size_t queue_size = psize + 1; // To be changed later -- equals grid dimension of request manager
  // size_t num_nodes = psize;      // To be changed later -- equals maximum multiplication factor
  queue_info *d_queue_space, *h_queue_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_queue_space, queue_size * sizeof(queue_info)));
  h_queue_space = (queue_info *)malloc(queue_size * sizeof(queue_info));
  memset(h_queue_space, 0, queue_size * sizeof(queue_info));
  for (size_t i = 0; i < queue_size; i++)
  {
    // CUDA_RUNTIME(hipMalloc((void **)&h_queue_space[i].nodes, num_nodes * sizeof(node)));
    // std::fill(h_queue_space[i].nodes, 0, num_nodes * sizeof(node));
    // std::fill(h_queue_space[i].nodes, h_queue_space[i].nodes + num_nodes, 0);
    h_queue_space[i].req_status.store(0, cuda::memory_order_release);
    h_queue_space[i].batch_size = 0;
    h_queue_space[i].id = (uint32_t)i;
  }
  CUDA_RUNTIME(hipMemcpy(d_queue_space, h_queue_space, queue_size * sizeof(queue_info), hipMemcpyHostToDevice));
  delete[] h_queue_space;

  // Create space for bound computation and branching
  work_info *d_work_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_work_space, queue_size * sizeof(work_info)));
  CUDA_RUNTIME(hipMemset((void *)d_work_space, 0, queue_size * sizeof(work_info)));

  // Create MPMC queue for handling heap requests
  queue_declare(request_queue, tickets, head, tail);
  queue_init(request_queue, tickets, head, tail, queue_size, dev_);

  // Create space for node_info and addresses
  size_t max_node_length = min(MAX_TOKENS, psize); // To be changed later -- equals problem size
  uint max_workers = psize + 1;
  node_info *d_node_space;

  uint *d_address_space; // To store dequeued addresses
  CUDA_RUNTIME(hipMallocManaged((void **)&d_address_space, max_workers * max_node_length * sizeof(uint)));
  CUDA_RUNTIME(hipMemset((void *)d_address_space, 0, max_workers * max_node_length * sizeof(uint)));

  // uint memory_queue_len = MAX_HEAP_SIZE;
  // Get memory queue length based on available memory
  // size_t free, total;
  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %f %", ((total - free) * 1.0) / total * 100);
  size_t memory_queue_len = (free * 0.95) / (sizeof(node_info) + sizeof(node)); // Keeping 5% headroom
  Log(info, "Memory queue length: %lu", memory_queue_len);

  CUDA_RUNTIME(hipMalloc((void **)&d_node_space, memory_queue_len * sizeof(node_info)));
  CUDA_RUNTIME(hipMemset((void *)d_node_space, 0, memory_queue_len * sizeof(node_info)));

  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %f %", ((total - free) * 1.0) / total * 100);
  // Create BHEAP on device
  BHEAP<node> d_bheap = BHEAP<node>(memory_queue_len, dev_);

  // Create bnb-stats object on device
  bnb_stats *stats;
  CUDA_RUNTIME(hipMallocManaged((void **)&stats, sizeof(bnb_stats)));
  stats->nodes_explored = 1; // for root node
  stats->nodes_pruned_incumbent = 0;
  stats->nodes_pruned_infeasible = 0;

  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %f %", ((total - free) * 1.0) / total * 100);
  // Create MPMC queue for handling memory requests
  queue_declare(memory_queue, tickets, head, tail);
  queue_init(memory_queue, tickets, head, tail, memory_queue_len, dev_);

  // Populate memory queue and node_space IDs
  execKernel(fill_memory_queue, memory_queue_len, 32, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), d_node_space,
             memory_queue_len);
  execKernel(check_queue_global, 1, 1, dev_, false, queue_caller(memory_queue, tickets, head, tail),
             memory_queue_len);

  // Frist kernel to create L1 nodes
  execKernel(initial_branching, 2, 32, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), memory_queue_len,
             d_address_space, d_node_space,
             d_problem_info, max_node_length,
             queue_caller(request_queue, tickets, head, tail), queue_size,
             d_queue_space, d_work_space, d_bheap,
             UB);

  execKernel(branch_n_bound, psize + 1, 32, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), memory_queue_len,
             d_address_space, d_node_space,
             d_problem_info, max_node_length,
             queue_caller(request_queue, tickets, head, tail), queue_size,
             d_queue_space, d_work_space, d_bheap,
             UB, stats);

  printf("\n");

  // Get exit code
  ExitCode exit_code, *d_exit_code;
  CUDA_RUNTIME(hipMalloc((void **)&d_exit_code, sizeof(ExitCode)));
  execKernel(get_exit_code, 1, 1, dev_, false, d_exit_code);
  CUDA_RUNTIME(hipMemcpy(&exit_code, d_exit_code, sizeof(ExitCode), hipMemcpyDeviceToHost));
  CUDA_RUNTIME(hipFree(d_exit_code));

  d_bheap.print_size();
  Log(info, "Max heap size during execution: %lu", d_bheap.d_max_size[0]);
  Log(info, "Nodes Explored: %u, Incumbant: %u, Infeasible: %u", stats->nodes_explored, stats->nodes_pruned_incumbent, stats->nodes_pruned_infeasible);
  Log(info, "Total time taken: %f sec", t.elapsed());

  // Free device memory
  d_bheap.free_memory();
  CUDA_RUNTIME(hipFree(d_queue_space));
  CUDA_RUNTIME(hipFree(d_node_space));
  CUDA_RUNTIME(hipFree(d_address_space));
  CUDA_RUNTIME(hipFree(d_work_space));
  CUDA_RUNTIME(hipFree(stats));
  CUDA_RUNTIME(hipFree(d_problem_info->costs));
  CUDA_RUNTIME(hipFree(d_problem_info->weights));
  CUDA_RUNTIME(hipFree(d_problem_info->budgets));
  CUDA_RUNTIME(hipFree(d_problem_info));

  delete[] h_problem_info->costs;
  delete[] h_problem_info->weights;
  delete[] h_problem_info->budgets;
  delete[] h_problem_info;

  queue_free(request_queue, tickets, head, tail);
  queue_free(memory_queue, tickets, head, tail);

  return int(exit_code);
}

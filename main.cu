#include <stdio.h>
#include <cmath>
#include <vector>
#include "heap/bheap.cuh"
#include "utils/logger.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/timer.h"
#include "queue/queue.cuh"
#include "request_manager.cuh"
#include "memory_manager.cuh"
#include "defs.cuh"

#include "LAP/config.h"
#include "LAP/cost_generator.h"
#include "LAP/device_utils.cuh"
#include "LAP/Hung_lap.cuh"
#include "LAP/lap_kernels.cuh"

int main(int argc, char **argv)
{
  Log(debug, "Starting program");
  Config config = parseArgs(argc, argv);
  printConfig(config);
  int dev_ = config.deviceId;
  CUDA_RUNTIME(hipDeviceReset());
  CUDA_RUNTIME(hipSetDevice(dev_));

  typedef uint data;
  double time;
  Timer t;
  data *h_costs = generate_cost<data>(config, config.seed);
  time = t.elapsed();
  Log(debug, "cost generation time %f s", time);
  t.reset();

  LAP<data> *lap = new LAP<data>(h_costs, config.user_n, dev_);
  Log(debug, "LAP object generated succesfully");
  lap->solve();
  time = t.elapsed();
  data UB = lap->objective;

  Log(debug, "LAP solved succesfully, objective %u\n", (uint)UB);

  /*
  INSTRUCTIONS ilist;
  ilist.populate_ins_from_file(fptr);
  ilist.print();
  d_instruction *d_ilist = ilist.to_device_array();

  // create BHEAP on device
  BHEAP<node> d_bheap;
  CUDA_RUNTIME(hipMalloc((void **)&d_bheap.d_heap, MAX_HEAP_SIZE * sizeof(node)));
  CUDA_RUNTIME(hipMalloc((void **)&d_bheap.d_size, sizeof(size_t)));

  size_t ins_len = ilist.tasks.size();
  const size_t max_batch = ilist.get_max_batch_size();

  // Create space for queue
  size_t queue_size = 100; // To be changed later -- equals grid dimension of request manager
  size_t num_nodes = 100;  // To be changed later -- equals maximum multiplication factor
  queue_info *d_queue_space, *h_queue_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_queue_space, queue_size * sizeof(queue_info)));
  h_queue_space = (queue_info *)malloc(queue_size * sizeof(queue_info));
  for (size_t i = 0; i < queue_size; i++)
  {
    CUDA_RUNTIME(hipMalloc((void **)&h_queue_space[i].values, num_nodes * sizeof(node)));
    CUDA_RUNTIME(hipMemset(h_queue_space[i].values, 0, num_nodes * sizeof(node)));
    h_queue_space[i].batch_size = 0;
    h_queue_space[i].already_occupied = int(false);
    h_queue_space[i].id = (uint32_t)i;
  }
  CUDA_RUNTIME(hipMemcpy(d_queue_space, h_queue_space, queue_size * sizeof(queue_info), hipMemcpyHostToDevice));
  delete[] h_queue_space;

  // Create space for node_info and addresses
  size_t max_node_length = MAX_TOKENS; // To be changed later -- equals problem size
  uint memory_queue_len = MAX_HEAP_SIZE;
  uint max_workers = 3;
  nodetype **d_node_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_node_space, memory_queue_len * max_node_length * sizeof(nodetype)));
  CUDA_RUNTIME(hipMemset((void *)d_node_space, 0, memory_queue_len * max_node_length * sizeof(nodetype)));

  uint *d_address_space;
  CUDA_RUNTIME(hipMallocManaged((void **)&d_address_space, max_workers * max_node_length * sizeof(uint)));
  CUDA_RUNTIME(hipMemset((void *)d_address_space, 0, max_workers * max_node_length * sizeof(uint)));

  // Create MPMC queue for handling memory requests
  cuda::atomic<uint32_t, cuda::thread_scope_device> *work_ready_memory = nullptr;
  queue_declare(memory_queue, tickets, head, tail);
  queue_init(memory_queue, tickets, head, tail, memory_queue_len, dev_);
  CUDA_RUNTIME(hipMalloc((void **)&work_ready_memory, memory_queue_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));
  CUDA_RUNTIME(hipMemset((void *)work_ready_memory, 0, memory_queue_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));

  // Populate memory queue
  execKernel(fill_memory_queue, memory_queue_len, 32, dev_, true, queue_caller(memory_queue, tickets, head, tail),
             memory_queue_len, work_ready_memory);
  execKernel(check_queue_global, 1, 1, dev_, true, queue_caller(memory_queue, tickets, head, tail),
             memory_queue_len, work_ready_memory);
  // execKernel(get_queue_length_global, 1, 1, dev_, true, queue_caller(memory_queue, tickets, head, tail));

  // execKernel(get_memory_global, max_workers, 32, dev_, true, queue_caller(memory_queue, tickets, head, tail),
  //            memory_queue_len, max_node_length, d_address_space, work_ready_memory);

  execKernel(memory_test, max_workers, 32, dev_, true, queue_caller(memory_queue, tickets, head, tail),
             memory_queue_len, max_node_length, d_address_space, d_address_space, work_ready_memory);

  execKernel(get_queue_length_global, 1, 1, dev_, true, queue_caller(memory_queue, tickets, head, tail));

  // print d_address_space directly from unified memory
  for (size_t i = 0; i < max_workers; i++)
  {
    printf("Worker %d: \n", i);
    for (size_t j = 0; j < max_node_length; j++)
    {
      printf("%d, ", d_address_space[i * MAX_TOKENS + j]);
    }
    printf("\n\n");
  }

  // execKernel(free_memory_global, max_workers, 32, dev_, true, queue_caller(memory_queue, tickets, head, tail),
  //            memory_queue_len, d_address_space);
  // execKernel(get_queue_length_global, 1, 1, dev_, true, queue_caller(memory_queue, tickets, head, tail));

  // Create MPMC queue for handling heap requests
  cuda::atomic<uint32_t, cuda::thread_scope_device> *work_ready_requests = nullptr;
  queue_declare(request_queue, tickets, head, tail);
  queue_init(request_queue, tickets, head, tail, ins_len, dev_);
  CUDA_RUNTIME(hipMalloc((void **)&work_ready_requests, ins_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));
  CUDA_RUNTIME(hipMemset((void *)work_ready_requests, 0, ins_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));

  ins_len = ilist.tasks.size();

  execKernel((request_manager<node>), max_workers + 1, 32, dev_, true,
             d_ilist, ins_len, queue_caller(request_queue, tickets, head, tail), work_ready_requests,
             queue_size, d_bheap, d_queue_space);
  d_bheap.print();
  // Free device memory
  CUDA_RUNTIME(hipFree(d_bheap.d_heap));
  CUDA_RUNTIME(hipFree(d_ilist));
  CUDA_RUNTIME(hipFree(d_bheap.d_size));
  CUDA_RUNTIME(hipFree(d_queue_space));
  CUDA_RUNTIME(hipFree(work_ready_requests));
  CUDA_RUNTIME(hipFree(work_ready_memory));
  queue_free(request_queue, tickets, head, tail);
  queue_free(memory_queue, tickets, head, tail);
  */

  return 0;
}
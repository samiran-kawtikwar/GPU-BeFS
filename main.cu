#include <stdio.h>
#include <cmath>
#include <vector>
#include "utils/logger.cuh"
#include "instr_parser.cuh"
#include "heap/bheap.cuh"
#include "utils/cuda_utils.cuh"
#include "queue/queue.cuh"
#include "request_manager.cuh"
#include "memory_manager.cuh"
#include "defs.cuh"

int main(int argc, char **argv)
{
  Log(debug, "Starting program");
  uint dev_ = 0;
  CUDA_RUNTIME(hipSetDevice(dev_));
  const char *fileName = argv[1];
  Log(debug, "File name: %s", fileName);

  FILE *fptr = fopen(fileName, "r");
  if (fptr == NULL)
  {
    Log(error, "%s file failed to open.", fileName);
    exit(-1);
  }
  INSTRUCTIONS ilist;
  ilist.populate_ins_from_file(fptr);
  ilist.print();
  d_instruction *d_ilist = ilist.to_device_array();

  // create BHEAP on device
  BHEAP<node> d_bheap;
  CUDA_RUNTIME(hipMalloc((void **)&d_bheap.d_heap, MAX_HEAP_SIZE * sizeof(node)));
  CUDA_RUNTIME(hipMalloc((void **)&d_bheap.d_size, sizeof(size_t)));

  size_t ins_len = ilist.tasks.size();
  const size_t max_batch = ilist.get_max_batch_size();

  // Create space for queue
  size_t queue_size = 100; // To be changed later -- equals grid dimension of request manager
  size_t num_nodes = 100;  // To be changed later -- equals maximum multiplication factor
  queue_info *d_queue_space, *h_queue_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_queue_space, queue_size * sizeof(queue_info)));
  h_queue_space = (queue_info *)malloc(queue_size * sizeof(queue_info));
  for (size_t i = 0; i < queue_size; i++)
  {
    CUDA_RUNTIME(hipMalloc((void **)&h_queue_space[i].values, num_nodes * sizeof(node)));
    CUDA_RUNTIME(hipMemset(h_queue_space[i].values, 0, num_nodes * sizeof(node)));
    h_queue_space[i].batch_size = 0;
    h_queue_space[i].already_occupied = int(false);
    h_queue_space[i].id = (uint32_t)i;
  }
  CUDA_RUNTIME(hipMemcpy(d_queue_space, h_queue_space, queue_size * sizeof(queue_info), hipMemcpyHostToDevice));
  delete[] h_queue_space;

  // Create space for node_info
  size_t max_node_length = 100; // To be changed later -- equals problem size
  uint memory_queue_len = MAX_HEAP_SIZE;
  nodetype **d_node_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_node_space, memory_queue_len * max_node_length * sizeof(nodetype)));
  CUDA_RUNTIME(hipMemset(d_node_space, 0, memory_queue_len * max_node_length * sizeof(nodetype)));

  // Create MPMC queue for handling memory requests
  cuda::atomic<uint32_t, cuda::thread_scope_device> *work_ready_memory = nullptr;
  queue_declare(memory_queue, tickets, head, tail);
  queue_init(memory_queue, tickets, head, tail, memory_queue_len, dev_);
  CUDA_RUNTIME(hipMalloc((void **)&work_ready_memory, memory_queue_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));
  CUDA_RUNTIME(hipMemset((void *)work_ready_memory, 0, memory_queue_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));

  // Populate memory queue
  execKernel(populate_memory_queue, memory_queue_len, 32, dev_, true, queue_caller(memory_queue, tickets, head, tail), memory_queue_len, work_ready_memory);
  // execKernel(check_queue, 1, 1, dev_, true, queue_caller(memory_queue, tickets, head, tail), work_ready_memory, memory_queue_len);

  // Create MPMC queue for handling heap requests
  cuda::atomic<uint32_t, cuda::thread_scope_device> *work_ready_requests = nullptr;
  queue_declare(request_queue, tickets, head, tail);
  queue_init(request_queue, tickets, head, tail, ins_len, dev_);
  CUDA_RUNTIME(hipMalloc((void **)&work_ready_requests, ins_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));
  CUDA_RUNTIME(hipMemset((void *)work_ready_requests, 0, ins_len * sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>)));

  ins_len = ilist.tasks.size();

  execKernel((request_manager<node>), 3 + 1, 32, dev_, true,
             d_ilist, ins_len, queue_caller(request_queue, tickets, head, tail), work_ready_requests,
             queue_size, d_bheap, d_queue_space);
  d_bheap.print();
  // Free device memory
  CUDA_RUNTIME(hipFree(d_bheap.d_heap));
  CUDA_RUNTIME(hipFree(d_ilist));
  CUDA_RUNTIME(hipFree(d_bheap.d_size));
  CUDA_RUNTIME(hipFree(d_queue_space));
  CUDA_RUNTIME(hipFree(work_ready_requests));
  CUDA_RUNTIME(hipFree(work_ready_memory));
  queue_free(request_queue, tickets, head, tail);
  queue_free(memory_queue, tickets, head, tail);

  return 0;
}
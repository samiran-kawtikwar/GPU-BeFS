#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <vector>
#include "heap/bheap.cuh"
#include "utils/logger.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/timer.h"
#include "queue/queue.cuh"
#include "request_manager.cuh"
#include "memory_manager.cuh"
#include "defs.cuh"
#include "branch.cuh"

#include "RCAP/config.h"
#include "RCAP/cost_generator.h"
#include "RCAP/gurobi_solver.h"
#include "RCAP/subgrad_solver.cuh"

#include "cudaProfiler.h"
#include "hip/hip_runtime_api.h"

__global__ void get_exit_code(ExitCode *ec)
{

  ec[0] = opt_reached.load(cuda::memory_order_consume)      ? ExitCode::OPTIMAL
          : heap_overflow.load(cuda::memory_order_consume)  ? ExitCode::HEAP_FULL
          : heap_underflow.load(cuda::memory_order_consume) ? ExitCode::INFEASIBLE
                                                            : ExitCode::UNKNOWN_ERROR;
}

int main(int argc, char **argv)
{
  Log(info, "Starting program");
  Config config = parseArgs(argc, argv);
  int dev_ = config.deviceId;
  CUDA_RUNTIME(hipSetDevice(dev_));
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev_);
  problem_info *pinfo = generate_problem<cost_type>(config, config.seed);
  // print(pinfo, true, true, false);
  printConfig(config);

  Timer t = Timer();
  uint psize = config.user_n, ncommodities = config.user_ncommodities;
  // Solve RCAP for getting UB (Can be taken from a library if not interesting in using gurobi)
  cost_type UB = solve_with_gurobi<cost_type, weight_type>(pinfo->costs, pinfo->weights, pinfo->budgets, psize, ncommodities);
  Log(info, "RCAP solved with GUROBI: objective %u\n", (uint)UB);
  Log(info, "Time taken by Gurobi: %f sec", t.elapsed());

  opt_reached.store(false, cuda::memory_order_release);
  heap_overflow.store(false, cuda::memory_order_release);
  heap_underflow.store(false, cuda::memory_order_release);

  Log(debug, "Solving RCAP with Branching");
  t.reset();

  // Create space for queue
  // Find max concurrent blocks for the branch_n_bound kernel

  int nw1, nb1;
  hipOccupancyMaxPotentialBlockSize(&nw1, &nb1, branch_n_bound, 0, 0);
  Log(info, "Max potential block size: %d", nb1);
  Log(info, "Max potential grid size: %d", nw1);

  assert(nb1 >= BlockSize);
  int nworkers; // equals grid dimension of request manager
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&nworkers, branch_n_bound, BlockSize, 0);
  Log(debug, "Max concurrent blocks per SM: %d", nworkers);
  nworkers *= deviceProp.multiProcessorCount;

  // Create space for bound computation storing and branching
  Log(debug, "Creating scratch space for workers");
  worker_info *d_worker_space = nullptr; // managed by each worker
  Log(debug, "Allocating space for %u workers with psize %u", nworkers, psize);
  worker_info::allocate_all(d_worker_space, nworkers, psize);

  int nsubworkers = BlockSize / TileSize;
  Log(debug, "Creating space for subgrad solver");
  subgrad_space *d_subgrad_space = nullptr; // managed by each subworker
  CUDA_RUNTIME(hipMallocManaged((void **)&d_subgrad_space, nsubworkers * nworkers * sizeof(subgrad_space)));
  d_subgrad_space->allocate(psize, ncommodities, nsubworkers * nworkers, dev_);

  Log(debug, "Creating space for request queue");
  queue_info *d_queue_space = nullptr;
  queue_info::allocate_all(d_queue_space, nworkers);

  // Create MPMC queue for handling heap requests
  queue_declare(request_queue, tickets, head, tail);
  queue_init(request_queue, tickets, head, tail, nworkers, dev_);

  // Get memory queue length based on available memory
  size_t free, total;
  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %.3f%%", ((total - free) * 1.0) / total * 100);
  size_t memory_queue_weight = (sizeof(node_info) + sizeof(node) + psize * sizeof(int) + sizeof(queue_type) + sizeof(cuda::atomic<uint32_t, cuda::thread_scope_device>));
  size_t memory_queue_len = (free * 0.95) / memory_queue_weight; // Keeping 5% headroom
  Log(info, "Memory queue length: %lu", memory_queue_len);

  // Create space for node_info
  node_info *d_node_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_node_space, memory_queue_len * sizeof(node_info)));
  CUDA_RUNTIME(hipMemset((void *)d_node_space, 0, memory_queue_len * sizeof(node_info)));
  // space for fixed assignments in node_info
  int *d_fixed_assignment_space;
  CUDA_RUNTIME(hipMalloc((void **)&d_fixed_assignment_space, memory_queue_len * psize * sizeof(int)));
  CUDA_RUNTIME(hipMemset((void *)d_fixed_assignment_space, 0, memory_queue_len * psize * sizeof(int)));

  // Set fixed assignment pointers to d_fixed_assignment_space
  uint block_dimension = 1024;
  uint grid_dimension = min(size_t(deviceProp.maxGridSize[0]), (memory_queue_len - 1) / block_dimension + 1);
  execKernel(set_fixed_assignment_pointers, grid_dimension, block_dimension, dev_, true,
             d_node_space, d_fixed_assignment_space, psize, memory_queue_len);

  // create space for hold_status
  bool *d_hold_status; // Managed by Workers
  CUDA_RUNTIME(hipMalloc((void **)&d_hold_status, nworkers * sizeof(bool)));
  CUDA_RUNTIME(hipMemset((void *)d_hold_status, 0, nworkers * sizeof(bool)));

  // Create BHEAP on device
  BHEAP<node> d_bheap = BHEAP<node>(memory_queue_len, dev_);

  // Create bnb-stats object on device
  bnb_stats *stats;
  CUDA_RUNTIME(hipMallocManaged((void **)&stats, sizeof(bnb_stats)));
  stats->initialize();

  // Create MPMC queue for handling memory requests
  queue_declare(memory_queue, tickets, head, tail);
  queue_init(memory_queue, tickets, head, tail, memory_queue_len, dev_);

  CUDA_RUNTIME(hipMemGetInfo(&free, &total));
  Log(info, "Occupied memory: %.3f%%", ((total - free) * 1.0) / total * 100);

#ifdef TIMER
  allocateCounters(&counters, nworkers);
  Log(debug, "Allocated regular counters");
  allocateCounters(&lap_counters, nworkers);
  Log(debug, "Allocated lap counters");
#endif

  // Populate memory queue and node_space IDs
  execKernel(fill_memory_queue, grid_dimension, block_dimension, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), d_node_space,
             memory_queue_len);

  Log(warn, "TileSize: %u", TileSize);
  // Frist kernel to create L1 nodes
  execKernel(initial_branching, 2, BlockSize, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), memory_queue_len,
             d_node_space, pinfo,
             queue_caller(request_queue, tickets, head, tail), nworkers,
             d_queue_space, d_worker_space, d_bheap,
             d_hold_status, UB);
  hipProfilerStart();
  execKernel(branch_n_bound, nworkers, BlockSize, dev_, true,
             queue_caller(memory_queue, tickets, head, tail), memory_queue_len,
             d_node_space, d_subgrad_space, pinfo,
             queue_caller(request_queue, tickets, head, tail), nworkers,
             d_queue_space, d_worker_space, d_bheap,
             d_hold_status,
             UB, stats);
  hipProfilerStop();
  Log(warn, "BnB Terminated");

#ifdef TIMER
  printCounters(counters, false);
  // printCounters(lap_counters, false);
  freeCounters(counters);
  freeCounters(lap_counters);
#endif

  // Get exit code
  ExitCode exit_code, *d_exit_code;
  CUDA_RUNTIME(hipMalloc((void **)&d_exit_code, sizeof(ExitCode)));
  execKernel(get_exit_code, 1, 1, dev_, false, d_exit_code);
  CUDA_RUNTIME(hipMemcpy(&exit_code, d_exit_code, sizeof(ExitCode), hipMemcpyDeviceToHost));
  CUDA_RUNTIME(hipFree(d_exit_code));

  d_bheap.print_size();
  Log(info, "Max heap size during execution: %lu", d_bheap.d_max_size[0]);
  Log(info, "Nodes Explored: %u, Incumbent: %u, Infeasible: %u", stats->nodes_explored, stats->nodes_pruned_incumbent, stats->nodes_pruned_infeasible);
  Log(info, "Total time taken: %f sec", t.elapsed());

  // Free device memory
  d_bheap.free_memory();
  CUDA_RUNTIME(hipFree(d_node_space));
  CUDA_RUNTIME(hipFree(d_fixed_assignment_space));
  CUDA_RUNTIME(hipFree(stats));
  CUDA_RUNTIME(hipFree(d_hold_status));

  worker_info::free_all(d_worker_space, nworkers);
  queue_info::free_all(d_queue_space);
  d_subgrad_space->clear();
  CUDA_RUNTIME(hipFree(d_subgrad_space));
  CUDA_RUNTIME(hipFree(pinfo));

  queue_free(request_queue, tickets, head, tail);
  queue_free(memory_queue, tickets, head, tail);

  // print exit code message and return
  Log(info, "Exit code: %s", ExitCode_text[exit_code]);
  return int(exit_code);
}
